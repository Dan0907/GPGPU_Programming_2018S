#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "SyncedMemory.h"

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

const int W = 40;
const int H = 12;

__global__ void Draw(char *frame) {
	// TODO: draw more complex things here
	// Do not just submit the original file provided by the TA!
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (y < H and x < W) {
		char c;
		if (x == W-1) {
			c = y == H-1 ? '\0' : '\n';
		} else if (y == 0 or y == H-1 or x == 0 or x == W-2) {
			c = ':';
		} else if ((x == 10 or x == 28) and y >= 4 and y <= 8) {
			c = 'o';
		} else if ((y == 4 or y == 8) and x > 10 and x < 28) {
			c = 'o';
		} else if (x == 19 and y >= 2 and y <=11) {
			c = 'o';
		} else {
			c = ' ';
		}
		frame[y*W+x] = c;
	}
}

int main(int argc, char **argv)
{
	MemoryBuffer<char> frame(W*H);
	auto frame_smem = frame.CreateSync(W*H);
	CHECK;

	Draw<<<dim3((W-1)/16+1,(H-1)/12+1), dim3(16,12)>>>(frame_smem.get_gpu_wo());
	CHECK;

	puts(frame_smem.get_cpu_ro());
	CHECK;
	return 0;
}
