#include "hip/hip_runtime.h"
#include "counting.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#define BLOCK_SIZE 256

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

struct IsChar {
	__host__ __device__
	int operator()(const char x)
	{
		return x != '\n' ? 1 : 0;
	}
};
void CountPosition1(const char *text, int *pos, int text_size)
{
	thrust::device_ptr<const char> data_ptr(text);
	thrust::device_ptr<int> result_ptr(pos);
	thrust::transform(thrust::device, data_ptr, data_ptr + text_size,
		result_ptr, IsChar());
	thrust::inclusive_scan_by_key(thrust::device, result_ptr,
		result_ptr + text_size, result_ptr, result_ptr);
}

__global__ void scan(int *pos, int n)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n)
		return;
	if (i >= pos[i])
		pos[i] += pos[i - pos[i]];
}

__global__ void init(const char *text, int *pos, int n)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n)
		return;
	if (text[i] != '\n')
		pos[i] = 1;
}
__global__ void slow(int *pos, int n)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n)
		return;
	int j;
	for (j = 0; i - j >= 0 &&  pos[i - j] != 0; j++);
	pos[i] = j;
}

void CountPosition2(const char *text, int *pos, int text_size)
{
	int i;
	int grid_size = CeilDiv(text_size, BLOCK_SIZE);
	init<<<grid_size, BLOCK_SIZE>>>(text, pos, text_size);
	//slow<<<grid_size, BLOCK_SIZE>>>(pos, text_size);
	for (i = 0; i < 9; i++)
		scan<<<grid_size, BLOCK_SIZE>>>(pos, text_size);
}
