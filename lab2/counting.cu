#include "hip/hip_runtime.h"
#include "counting.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#define BLOCK_SIZE 256

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

struct IsChar {
	__host__ __device__
	int operator()(const char x)
	{
		return x != '\n' ? 1 : 0;
	}
};
void CountPosition1(const char *text, int *pos, int text_size)
{
	thrust::device_ptr<const char> data_ptr(text);
	thrust::device_ptr<int> result_ptr(pos);
	thrust::transform(thrust::device, data_ptr, data_ptr + text_size,
		result_ptr, IsChar());
	thrust::inclusive_scan_by_key(thrust::device, result_ptr,
		result_ptr + text_size, result_ptr, result_ptr);
}

__global__ void scan(int *res, int n, int shift_num, int dir)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n)
		return;
	if ((i >> shift_num) % 2 == dir && i >= res[i]) {
		res[i] += res[i - res[i]];
	}
}

__global__ void init(const char *text, int *res, int n)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n)
		return;
	if (text[i] != '\n')
		res[i] = 1;
}

void CountPosition2(const char *text, int *pos, int text_size)
{
	int i;
	int grid_size = CeilDiv(text_size, BLOCK_SIZE);
	init<<<grid_size, BLOCK_SIZE>>>(text, pos, text_size);
	for (i = 0; i <= 8; i++) {
		scan<<<grid_size, BLOCK_SIZE>>>(pos, text_size, i, 0);
		scan<<<grid_size, BLOCK_SIZE>>>(pos, text_size, i, 1);
	}
}
